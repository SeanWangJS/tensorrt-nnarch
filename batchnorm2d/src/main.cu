#include "hip/hip_runtime.h"
// https://www.nvidia.cn/content/dam/en-zz/zh_cn/assets/webinars/oct16/Gary_TensorRT_GTCChina2019.pdf

#include <iostream>
#include <string>
#include <fstream>

#include <NvInfer.h>

using namespace nvinfer1;

class Logger : public nvinfer1::ILogger{
    
    void log(Severity severity, const char* msg) noexcept override {
        std::cout << msg << std::endl;
    }
    
};


nvinfer1::Weights loadWeight(const std::string& weightDir, const std::string& weightName) {
    std::string weightPath = weightDir + "/" + weightName + ".bin";
    std::ifstream file(weightPath, std::ios::binary|std::ios::ate);
    if(!file) {
        std::cerr << "Error opening file" << std::endl;
        return nvinfer1::Weights();
    }

    file.seekg(0, file.end);
    int length = file.tellg();
    file.seekg(0, file.beg);

    char* data = new char[length];
    file.read(data, length);

    nvinfer1::Weights weight;
    weight.type = nvinfer1::DataType::kFLOAT;
    weight.values = data;
    weight.count = length / sizeof(float);

    return weight;

}

nvinfer1::Weights generateRandomWeights(int count) {

    float* values = new float[count];

    for(int i = 0; i < count; i++) {
        float d = static_cast<float>(rand() / static_cast<float>(RAND_MAX));
        values[i] = d;
    }

    nvinfer1::Weights weights;
    weights.type = nvinfer1::DataType::kFLOAT;
    weights.values = values;
    weights.count = count;

    return weights;

}


void saveEngineData(nvinfer1::IHostMemory* engine, const std::string& filepath) {

    std::ofstream engineFile(filepath, std::ios::binary);
    if(!engineFile) {
        std::cerr << "Error opening file" << std::endl;
        return;
    }   

    engineFile.write(static_cast<const char*>(engine -> data()), engine -> size());

    engineFile.close();

}

int main(int argc, char** argv) {

    Logger logger;
    nvinfer1::IBuilder* builder = nvinfer1::createInferBuilder(logger);
    int flag = 1U << static_cast<int>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
    nvinfer1::INetworkDefinition* network = builder -> createNetworkV2(flag);

    int32_t numChannels = 64;
    nvinfer1::ITensor* input = network -> addInput("input", nvinfer1::DataType::kFLOAT, nvinfer1::Dims{4, {1, numChannels, 100, 100}});

    std::string weightDir = "../weights";
    
    nvinfer1::Weights alpha = loadWeight(weightDir, "weight");
    nvinfer1::Weights beta = loadWeight(weightDir, "bias");
    nvinfer1::Weights mean = loadWeight(weightDir, "running_mean");
    nvinfer1::Weights var = loadWeight(weightDir, "running_var");

    const float* a = static_cast<const float*>(alpha.values);
    const float* b = static_cast<const float*>(beta.values);
    const float* m = static_cast<const float*>(mean.values);
    const float* v = static_cast<const float*>(var.values);

    float* scale = new float[numChannels];
    float* shift = new float[numChannels];
    float* power = new float[numChannels];

    for(int i = 0; i < numChannels; i++) {
        scale[i] = a[i] / sqrt(v[i] + 1e-5);
        shift[i] = - a[i] / sqrt(v[i] + 1e-5) * m[i] + b[i];
        power[i] = 1.0;
    }

    nvinfer1::Weights scaleWeights;
    scaleWeights.type = nvinfer1::DataType::kFLOAT;
    scaleWeights.values = scale;
    scaleWeights.count = numChannels;

    nvinfer1::Weights shiftWeights;
    shiftWeights.type = nvinfer1::DataType::kFLOAT;
    shiftWeights.values = shift;
    shiftWeights.count = numChannels;

    nvinfer1::Weights powerWeights;
    powerWeights.type = nvinfer1::DataType::kFLOAT;
    powerWeights.values = power;
    powerWeights.count = numChannels;

    nvinfer1::IScaleLayer* bnLayer = network -> addScaleNd(*input, 
                                                           nvinfer1::ScaleMode::kCHANNEL, 
                                                           shiftWeights, 
                                                           scaleWeights, 
                                                           powerWeights, 
                                                           1);

    bnLayer->getOutput(0)->setName("output");

    network -> markOutput(*bnLayer->getOutput(0));

    nvinfer1::IBuilderConfig* config = builder -> createBuilderConfig();
    config -> setMaxWorkspaceSize(1 << 20);
    nvinfer1::IHostMemory* engine = builder -> buildSerializedNetwork(*network, *config);

    std::string savePath = "../model.engine";

    saveEngineData(engine, savePath);

    return 0;

    
}