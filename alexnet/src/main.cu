#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <random>
#include <map>
#include <string>
#include <iostream>
#include <fstream>
#include <tuple>

#include <NvInfer.h>

#include "network.hpp"

using namespace nvinfer1;
using namespace nnarch;

nvinfer1::Weights generateRandomWeights(int count) {

    float* values = new float[count];

    for(int i = 0; i < count; i++) {
        float d = static_cast<float>(rand() / static_cast<float>(RAND_MAX));
        values[i] = d;
    }

    nvinfer1::Weights weights;
    weights.type = nvinfer1::DataType::kFLOAT;
    weights.values = values;
    weights.count = count;

    return weights;

}

nvinfer1::Weights loadWeight(const std::string& weightDir, const std::string& weightName) {
    std::string weightPath = weightDir + "/" + weightName + ".bin";
    std::ifstream file(weightPath, std::ios::binary|std::ios::ate);
    if(!file) {
        std::cerr << "Error opening file" << std::endl;
        return nvinfer1::Weights();
    }

    file.seekg(0, file.end);
    int length = file.tellg();
    file.seekg(0, file.beg);

    char* data = new char[length];
    file.read(data, length);

    nvinfer1::Weights weight;
    weight.type = nvinfer1::DataType::kFLOAT;
    weight.values = data;
    weight.count = length / sizeof(float);

    return weight;

}

class Logger : public nvinfer1::ILogger{
    
    void log(Severity severity, const char* msg) noexcept override {
        std::cout << msg << std::endl;
    }
    
};

void saveEngineData(nvinfer1::IHostMemory* engine, const std::string& filepath) {

    std::ofstream engineFile(filepath, std::ios::binary);
    if(!engineFile) {
        std::cerr << "Error opening file" << std::endl;
        return;
    }   

    engineFile.write(static_cast<const char*>(engine -> data()), engine -> size());

    engineFile.close();

}

void print(nvinfer1::Dims dims) {

    std::string str = "[";

    int MAX_DIMS = dims.MAX_DIMS;

    for(int i = 0; i < MAX_DIMS - 1; i++) {
        str += std::to_string(dims.d[i]) + ", ";
    }

    str += std::to_string(dims.d[MAX_DIMS - 1]) + "]";

    std::cout << str << std::endl;

}

int main(int argc, char** argv) {

    std::map<std::string, nvinfer1::Weights> weightsMap;

    int num_weights = 16;

    std::string weightNames[num_weights] = {
        "features.0.weight",
        "features.0.bias",
        "features.3.weight",
        "features.3.bias",
        "features.6.weight",
        "features.6.bias",
        "features.8.weight",
        "features.8.bias",
        "features.10.weight",
        "features.10.bias",
        "classifier.1.weight",
        "classifier.1.bias",
        "classifier.4.weight",
        "classifier.4.bias",
        "classifier.6.weight",
        "classifier.6.bias"
    };

    std::string weightDir = "../weights";
    for(int i = 0; i < num_weights; i++) {
        weightsMap[weightNames[i]] = loadWeight(weightDir, weightNames[i]);
    }

    Logger logger;
    nvinfer1::IBuilder* builder = nvinfer1::createInferBuilder(logger);
    int flag = 1U << static_cast<int>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
    nvinfer1::INetworkDefinition* network = builder -> createNetworkV2(flag);

    nnarch::NetworkApi api = nnarch::NetworkApi(network);

    nvinfer1::ITensor* input = network -> addInput("input", nvinfer1::DataType::kFLOAT, nvinfer1::Dims{4, {1, 3, 224, 224}});

    // Conv2d(3, 64, kernel_size=(11, 11), stride=(4, 4), padding=(2, 2))
    nvinfer1::IConvolutionLayer* conv1 = api.conv2d(input, 64, 11, weightsMap["features.0.weight"], weightsMap["features.0.bias"], 4, 2, "conv1");    
    
    std::cout << "Conv1 output dimensions: " << std::endl;
    print(conv1 -> getOutput(0) -> getDimensions());

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer* relu1 = api.relu(conv1 -> getOutput(0), "relu1");

    // MaxPool2d(kernel_size=3, stride=2, padding=0, dilation=1, ceil_mode=False)
    nvinfer1::IPoolingLayer* max_pool1 = api.maxpool2d(relu1 -> getOutput(0), 3, 2, 0, "max_pool1");

    std::cout << "MaxPool1 output dimensions: " << std::endl;
    print(max_pool1 -> getOutput(0) -> getDimensions());

    // Conv2d(64, 192, kernel_size=(5, 5), stride=(1, 1), padding=(2, 2))
    nvinfer1::IConvolutionLayer* conv2 = api.conv2d(max_pool1 -> getOutput(0), 192, 5, weightsMap["features.3.weight"], weightsMap["features.3.bias"], 1, 2, "conv2");

    std::cout << "Conv2 output dimensions: " << std::endl;
    print(conv2 -> getOutput(0) -> getDimensions());

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer* relu2 = api.relu(conv2 -> getOutput(0), "relu2");

    // MaxPool2d(kernel_size=3, stride=2, padding=0, dilation=1, ceil_mode=False)
    nvinfer1::IPoolingLayer* max_pool2 = api.maxpool2d(relu2 -> getOutput(0), 3, 2, 0, "max_pool2");

    std::cout << "MaxPool2 output dimensions: " << std::endl;
    print(max_pool2 -> getOutput(0) -> getDimensions());

    // Conv2d(192, 384, kernel_size=(3, 3), stride=(1, 1), padding=(1, 1))
    nvinfer1::IConvolutionLayer* conv3 = api.conv2d(max_pool2 -> getOutput(0), 384, 3, weightsMap["features.6.weight"], weightsMap["features.6.bias"], 1, 1, "conv3");

    std::cout << "Conv3 output dimensions: " << std::endl;
    print(conv3 -> getOutput(0) -> getDimensions());

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer* relu3 = api.relu(conv3 -> getOutput(0), "relu3");
    
    // Conv2d(384, 256, kernel_size=(3, 3), stride=(1, 1), padding=(1, 1))
    nvinfer1::IConvolutionLayer* conv4 = api.conv2d(relu3 -> getOutput(0), 256, 3, weightsMap["features.8.weight"], weightsMap["features.8.bias"], 1, 1, "conv4");

    std::cout << "Conv4 output dimensions: " << std::endl;
    print(conv4 -> getOutput(0) -> getDimensions());

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer* relu4 = api.relu(conv4 -> getOutput(0), "relu4");
    
    // Conv2d(256, 256, kernel_size=(3, 3), stride=(1, 1), padding=(1, 1))
    nvinfer1::IConvolutionLayer* conv5 = api.conv2d(relu4 -> getOutput(0), 256, 3, weightsMap["features.10.weight"], weightsMap["features.10.bias"], 1, 1, "conv5");

    std::cout << "Conv5 output dimensions: " << std::endl;
    print(conv5 -> getOutput(0) -> getDimensions());

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer* relu5 = api.relu(conv5 -> getOutput(0), "relu5");

    // MaxPool2d(kernel_size=3, stride=2, padding=0, dilation=1, ceil_mode=False)
    nvinfer1::IPoolingLayer* max_pool5 = api.maxpool2d(relu5 -> getOutput(0), 3, 2, 0, "max_pool5");

    std::cout << "MaxPool5 output dimensions: " << std::endl;
    print(max_pool5 -> getOutput(0) -> getDimensions());

    // AdaptiveAvgPool2d(output_size=(6, 6))
    nvinfer1::IPoolingLayer* avg_pool = api.avgpool2d(max_pool5 -> getOutput(0), 1, 1, 0, "avg_pool");

    nvinfer1::Dims dims = avg_pool -> getOutput(0) -> getDimensions();

    std::cout << "AvgPool output dimensions: " << std::endl;
    print(avg_pool -> getOutput(0) -> getDimensions());

    int32_t const batch = dims.d[0];
    int32_t const length = dims.d[1] * dims.d[2] * dims.d[3];
    nvinfer1::IShuffleLayer* reshapeLayer = api.reshape(avg_pool -> getOutput(0), Dims{2, {batch, length}}, "reshape");

    std::cout << "reshape output dimensions: " << std::endl;    
    print(reshapeLayer -> getOutput(0) -> getDimensions());

    // Linear(in_features=9216, out_features=4096, bias=True)
    nvinfer1::ILayer* linear1 = api.linear(reshapeLayer -> getOutput(0), 4096, weightsMap["classifier.1.weight"], weightsMap["classifier.1.bias"], "linear1");

    // ReLU(inplace=True)
    nvinfer1::IActivationLayer *relu6 = api.relu(linear1 -> getOutput(0), "relu6");

    // Linear(in_features=4096, out_features=4096, bias=True)
    nvinfer1::ILayer* linear2 = api.linear(relu6 -> getOutput(0), 4096, weightsMap["classifier.4.weight"], weightsMap["classifier.4.bias"], "linear2");
    
    // ReLU(inplace=True)
    nvinfer1::IActivationLayer *relu7 = api.relu(linear2 -> getOutput(0), "relu7");

    // Linear(in_features=4096, out_features=1000, bias=True)
    nvinfer1::ILayer* linear3 = api.linear(relu7 -> getOutput(0), 1000, weightsMap["classifier.6.weight"], weightsMap["classifier.6.bias"], "linear3");

    linear3 -> getOutput(0) -> setName("output");
    network -> markOutput(*linear3 -> getOutput(0));

    nvinfer1::IBuilderConfig* config = builder -> createBuilderConfig();
    config -> setMaxWorkspaceSize(1<<20);
    nvinfer1::IHostMemory* engine = builder -> buildSerializedNetwork(*network, *config);

    std::string savePath = "./model.engine";
    saveEngineData(engine, savePath);

    return 0;
}
